
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void saxpy(int n, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = x[i] + y[i];
}

void cuda_array_culc_add_float(float* x, float* y, int32_t N)
{
  float *d_x, *d_y;

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
}
